/*

	High Performance Computing
	Lab2

	Gabriel Bustamante
	Nicolas Zelada

*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#define C 1.0
#define DT 0.1
#define DD 2.0
#define F 20

// ./wave -N 256 -x 2 -y 2 -T 1001 -f salidas/aqui.raw

__global__ void olaInicial(float *H, float *H_1, int n){

	int i;
	i = blockDim.x*blockIdx.x + threadIdx.x;  // global index x (horizontal)
	// j = blockDim.y*blockIdx.y + threadIdx.y;  // global index y (vertical)

	if (i<n){
		for (int j = 0; j < n; j++){
			if(i==0 || j==0 || i==n-1 || j==n-1){
				H[i*n + j] = 0;
			}else{
				H[i*n + j] = H_1[i*n + j] + ((C*C)/2.0)*((DT*DT)/(DD*DD))*(H_1[(i+1)*n + j] + H_1[(i-1)*n + j] + H_1[i*n + (j-1)] + H_1[i*n + (j+1)] - 4*H_1[i*n + j]);
			}
		}
	}
}

__global__ void ola(float *H, float *H_1, float *H_2, int n){

	int i;
	i = blockDim.x*blockIdx.x + threadIdx.x;  // global index x (horizontal)
	// j = blockDim.y*blockIdx.y + threadIdx.y;  // global index y (vertical)

	if (i<n){
		for (int j = 0; j < n; j++){
			if(i==0 || j==0 || i==n-1 || j==n-1){
				H[i*n + j] = 0;
			}else{
				H[i*n + j] = 2*H_1[i*n + j] - H_2[i*n + j] + (C*C)*((DT*DT)/(DD*DD))*(H_1[(i+1)*n + j] + H_1[(i-1)*n + j] + H_1[i*n + (j-1)] + H_1[i*n + (j+1)] - 4*H_1[i*n + j]);
			}
		}
	}
}

__host__ int main(int argc, char* argv[]){

	char *nombreArchivo = NULL;
	int n = 0;
	int iteraciones = 0;
	int gridSize = 0; //x
	int blockSize = 0; //y

	// Parametros de entrada
	int c;
	while (((c = getopt(argc, argv, "N:x:y:T:f:")) != -1)){
		switch (c){
		case 'N': // tamano grilla
			n = atof(optarg);
			break;
		case 'x': // x
			gridSize = atof(optarg);
			break;
		case 'y': // y
			blockSize = atof(optarg);
			break;
		case 'T': // numero de pasos
			iteraciones = atof(optarg);
			break;
		case 'f': // archivo salida
			nombreArchivo = optarg;
			break;
		}
	}

	// Se reserva memoria para las variables del host
	float *H;
	float *H_1;
	float *H_2;
	H = (float*)malloc(n*n*sizeof(float));   // H actual
	H_1 = (float*)malloc(n*n*sizeof(float));	// H en t-1
	H_2 = (float*)malloc(n*n*sizeof(float));	// H en t-2

	// Se reserva memoria para las variables del device
	float *D_H;
	float *D_H_1;
	float *D_H_2;
	hipMallocManaged(&D_H, n*n*sizeof(float));
	hipMallocManaged(&D_H_1, n*n*sizeof(float));
	hipMallocManaged(&D_H_2, n*n*sizeof(float));

	int limiteInf = 0.4*n;	// limite inferior para inicio de la ola
	int limiteSup = 0.6*n;	// limite superior para inicio de la ola

	// Inicializar en 0 o F segun corresponda toda H
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			if(i>=limiteInf && i<=limiteSup && j>=limiteInf && j<=limiteSup){
				H_1[i*n + j] = F;
			}else{
				H_1[i*n + j] = 0;
			}
		}
	}

	// Copia de vectores del host al device
	hipMemcpy(D_H_1, H_1, n*n*sizeof(float), hipMemcpyHostToDevice);

	// Inicio de 
	for (int k = 0; k < iteraciones; k++){
		if(k==0){
			olaInicial<<<gridSize, blockSize>>>(D_H,D_H_1,n);
			// cudaDeviceSynchronize();
		}else{
			ola<<<gridSize, blockSize>>>(D_H,D_H_1,D_H_2,n);
			// cudaDeviceSynchronize();
		}

		// copiar todo device al host
		hipMemcpy(H, D_H, n*n*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(H_1, D_H_1, n*n*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(H_2, D_H_2, n*n*sizeof(float), hipMemcpyDeviceToHost);

		// Se guarda t-1 y t-2
		for (int i = 0; i < n; i++){
			for (int j = 0; j < n; j++){
				H_2[i * n + j] = H_1[i*n + j];
				H_1[i * n + j] = H[i*n + j];
			}
		}

		// copiar host al device
		hipMemcpy(D_H, H, n*n*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(D_H_1, H_1, n*n*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(D_H_2, H_2, n*n*sizeof(float), hipMemcpyDeviceToHost);
	}

	FILE *f = fopen(nombreArchivo,"w");
	fwrite(H,sizeof(float),n*n,f);
	fclose(f);

	// Liberacion de memoria
	free(H);
	free(H_1);
	free(H_2);
	hipFree(D_H);
	hipFree(D_H_1);
	hipFree(D_H_2);

	return 0;
}